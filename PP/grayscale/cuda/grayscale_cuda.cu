// nvcc grayscale_cuda.cu -o grayscale_cuda


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

typedef struct {
     unsigned char red, green, blue;
} PPMPixel;

typedef struct {
     int x, y;
     PPMPixel *data;
} PPMImage;

#define CREATOR "ParallelProgrammer"
#define RGB_COMPONENT_COLOR 255

static PPMImage *ReadPPM(const char *filename) {
     char buff[16];
     PPMImage *img;
     FILE *fp;
     int c, rgb_comp_color;

     // зареждане на PPM файла за четене
     fp = fopen(filename, "rb");
     if (!fp) {
          fprintf(stderr, "Unable to open file '%s'\n", filename);
          exit(1);
     }

     // четене на метаданните от него 
     if (!fgets(buff, sizeof(buff), fp)) {
          perror(filename);
          exit(1);
     }

     // проверка на метаданните
     if (buff[0] != 'P' || buff[1] != '6') {
          fprintf(stderr, "Invalid image format (must be 'P6')\n");
          exit(1);
     }

     // заделяне на памет
     img = (PPMImage *)malloc(sizeof(PPMImage));
     if (!img) {
          fprintf(stderr, "Unable to allocate memory\n");
          exit(1);
     }

     // проверка за коментари вътре в самото изображение
     c = getc(fp);
     while (c == '#') {
          while (getc(fp) != '\n') ;
          c = getc(fp);
     }

     ungetc(c, fp);
     // проверка на данните за размера на изображението
     if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
          fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
          exit(1);
     }

     // проверка на RGB компонента
     if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
          fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
          exit(1);
     }

     // проверка на размерността на RGB компонента
     if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
          fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
          exit(1);
     }

     while (fgetc(fp) != '\n') ;
     // заделяне на памет за информацията във всеки пиксел
     img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

     if (!img) {
          fprintf(stderr, "Unable to allocate memory\n");
          exit(1);
     }

     // зареждане на данните за всеки пиксел
     if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
          fprintf(stderr, "Error loading image '%s'\n", filename);
          exit(1);
     }

     fclose(fp);
     return img;
}
void WritePPM(const char *filename, PPMImage *img) {
     FILE *fp;
     // отваряне на файл в режим за писане
     fp = fopen(filename, "wb");
     if (!fp) {
          fprintf(stderr, "Unable to open file '%s'\n", filename);
          exit(1);
     }

     // записване на метаданни за типа на PPM изображението
     fprintf(fp, "P6\n");

     // запис на коментари
     fprintf(fp, "# Created by %s\n", CREATOR);

     // запис на размера на изображението
     fprintf(fp, "%d %d\n",img->x,img->y);

     // запис на размерността на RGB компонента
     fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

     // запис на данните за пикселите от изображението
     fwrite(img->data, 3 * img->x, img->y, fp);
     fclose(fp);
}

// kernel функция
__global__ void grayscale(int n, PPMPixel *source, PPMPixel *target) {
     int id = blockIdx.x*blockDim.x + threadIdx.x;
     if (id < n) {
          double f = 1;
          double l = 0.3 * source[id].red + 0.6 * source[id].green + 0.1 * source[id].blue;
          target[id].red = source[id].red + f * (l - source[id].red);
          target[id].green = source[id].green + f * (l - source[id].green);
          target[id].blue = source[id].blue + f * (l - source[id].blue);
     }
}

void ChangeColorPPM(PPMImage *img) {
     struct timeval tval_before, tval_after, tval_result;

     int N = img->x * img->y;
     PPMPixel *source, *d_source, *target, *d_target;

     // Заделяне на памет за входните и трансформираните данни в хоста
     source = (PPMPixel*)malloc(N*sizeof(PPMPixel));
     target = (PPMPixel*)malloc(N*sizeof(PPMPixel));

     // Заделяне на памет за входните и трансфомираните данни в cuda устройството
     hipMalloc(&d_source, N*sizeof(PPMPixel));
     hipMalloc(&d_target, N*sizeof(PPMPixel));

     // Измерване колко време отнема копирането на данните в устройството
     gettimeofday(&tval_before, NULL);

     // Копиране на данните от хоста в cuda устройството
     hipMemcpy(d_source, img->data, N*sizeof(PPMPixel), hipMemcpyHostToDevice);
     hipMemcpy(d_target, img->data, N*sizeof(PPMPixel), hipMemcpyHostToDevice);

     gettimeofday(&tval_after, NULL);
     
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на масивите в устройството\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     
     // Измерване колко време отнема изпънението на kernel функцията
     gettimeofday(&tval_before, NULL);

     // Изпълнение на kernel функцияа
     grayscale<<<(N+383)/384, 384>>>(N, d_source, d_target);
     gettimeofday(&tval_after, NULL);
     
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за изпълнението на kernel функцията\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     // printf("error: %s\n", cudaGetErrorString(cudaGetLastError()));

     gettimeofday(&tval_before, NULL);

     // Копиране на данните от устройството в хоста, за да можем да ги прочетем и запишем в изображението
     hipMemcpy(target, d_target, N*sizeof(PPMPixel), hipMemcpyDeviceToHost);
     gettimeofday(&tval_after, NULL);
     // Измерване колко време отнема копирането на данните в хоста
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на данните обратно в хоста\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     // printf("error: %s\n", cudaGetErrorString(cudaGetLastError()));

     gettimeofday(&tval_before, NULL);

     // Записване на получение трансформирани данни в структурата от данни, която се използва от записващата функция
     for (int i = 0; i < N; i++) {
          img->data[i] = target[i];
     }
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  %ld.%06ld   секунди за копиране на данните обратно в масива с пикселите\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
     // printf("error: %s\n", cudaGetErrorString(cudaGetLastError()));

     // Освобождаване на ресурси
     hipFree(d_source);
     hipFree(d_target);
     free(source);
     free(target);
}

int main(void) {
     PPMImage *image;
     struct timeval tval_before, tval_after, tval_result;

     gettimeofday(&tval_before, NULL);
     image = ReadPPM("image.ppm");
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("%ld.%06ld     секунди за четене на данните от изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

     gettimeofday(&tval_before, NULL);
     ChangeColorPPM(image);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("  --------\n");
     printf("  %ld.%06ld   секунди за обработка на данните от изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

     gettimeofday(&tval_before, NULL);
     WritePPM("grayscale_cuda_result.ppm", image);
     gettimeofday(&tval_after, NULL);
     timersub(&tval_after, &tval_before, &tval_result);
     printf("%ld.%06ld     секунди за запис на данните в изображението\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
}
